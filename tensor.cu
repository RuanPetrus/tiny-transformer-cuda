#include "hip/hip_runtime.h"
#include <assert.h>
#include <random>

#define TODO(message) do { fprintf(stderr, "%s:%d: TODO: %s\n", __FILE__, __LINE__, message); abort(); } while(0)
#define ERROR(...) do { fprintf(stderr, __VA_ARGS__); abort(); } while(0)
#define KEY(i3, i2, i1, d3, d2, d1) (i3*d2*d1 + i2*d1 + i1)

#define THREAD_PER_BLOCK_X (1 << 4)
#define THREAD_PER_BLOCK_Y (1 << 3)
#define THREAD_PER_BLOCK_Z (1 << 3)

__global__ void kernel_vector_add(float *c, float * a, float *b, int n) 
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < n) {
		c[id] = a[id] + b[id];
	}
}

__global__ void kernel_vector_mul(float *c, float * a, float *b, int n) 
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < n) {
		c[id] = a[id] * b[id];
	}
}

__global__ void kernel_vector_div(float *c, float * a, float *b, int n) 
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < n) {
		c[id] = a[id] / b[id];
	}
}

__global__ void kernel_vector_exp(float *c, float *a, int n) 
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < n) {
		c[id] = exp(a[id]);
	}
}

__global__ void kernel_mat_mul(float *c, float *a, float *b, int N, int M, int T)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < N && col < M) {
		float sum = 0;
		for (int k = 0; k < T; k++) {
			 sum += a[row*T + k] * b[k * M + col];
		}
		c[row*M + col] = sum;
	}
}

// Sum over axis
__global__ void kernel_sum(float *c, float *a, 
							int cd1, int cd2, int cd3, // c dimensions
							int ad1, int ad2, int ad3) // a dimensions
{
	int ci3 = blockIdx.z * blockDim.z + threadIdx.z;
	int ci2 = blockIdx.y * blockDim.y + threadIdx.y;
	int ci1 = blockIdx.x * blockDim.x + threadIdx.x;

	int kn = ad3 / cd3;
	int jn = ad2 / cd2;
	int in = ad1 / cd1;

	if (ci3 < cd3 && ci2 < cd2 && ci1 < cd1) {
		float sum = 0;
		for (int k = 0; k < kn; k++)
		for (int j = 0; j < jn; j++)
		for (int i = 0; i < in; i++) {
			int ai3 = ci3 + k;
			int ai2 = ci2 + j;
			int ai1 = ci1 + i;
			sum += a[KEY(ai3, ai2, ai1, ad3, ad2, ad1)];
		}
		c[KEY(ci3, ci2, ci1, cd3, cd2, cd1)] = sum;
	}
}

// Brodcast a to c
__global__ void kernel_broadcast(float *c, float *a, 
								 int cd1, int cd2, int cd3,
								 int ad1, int ad2, int ad3) 
{
	int ci3 = blockIdx.z * blockDim.z + threadIdx.z;
	int ci2 = blockIdx.y * blockDim.y + threadIdx.y;
	int ci1 = blockIdx.x * blockDim.x + threadIdx.x;

	int f1 = cd1 / ad1;
	int f2 = cd2 / ad2;
	int f3 = cd3 / ad3;

	int ai3 = ci3 / f3;
	int ai2 = ci2 / f2;
	int ai1 = ci1 / f1;

	if (ci3 < cd3 && ci2 < cd2 && ci1 < cd1) {
		// c[id3][id2][id1] = a[ai3][ai2][ai1];
		c[KEY(ci3, ci2, ci1, cd3, cd2, cd1)] = a[KEY(ai3, ai2, ai1, ad3, ad2, ad1)];
	}
}


static int TENSOR_GPU_LAST_SYNC = 0;

float random_normal_distribution_float()
{
	static std::random_device random_device{};
	static std::mt19937 random_generator{random_device()};
	static std::normal_distribution random_normal_distribution{0.0f, 1.0f};
	return random_normal_distribution(random_generator);
}

struct Tensor3 
{
	int d1, d2, d3;
	float *data; // Gpu data
	float *cpu_data;
	int sync;
};

Tensor3 tensor3_new(int d1, int d2, int d3, float *v)  // (d3, d2, d1)
{
	size_t size = d1 * d2 * d3  * sizeof(float);
	float *cpu_data =(float *) malloc(size);
	float *data; hipMalloc(&data, size);
	if (v != NULL) {
		hipMemcpy(data, v, size, hipMemcpyHostToDevice);
	}
	return {
		d1, d2, d3,
		data,
		cpu_data,
		-1
	};
}

Tensor3 tensor3_new(int d1, int d2, int d3) 
{
	return tensor3_new(d1, d2, d3, NULL);
}



Tensor3 tensor3_randn(int d1, int d2, int d3) 
{

	size_t size = d1 * d2 * d3  * sizeof(float);
	float *data =(float *) malloc(size);
	for (int i = 0; i < d1*d2*d3; i++) {
		data[i] = random_normal_distribution_float();
	}
	return tensor3_new(d1, d2, d3, data);
}

void tensor3_sync(Tensor3 t) 
{
	if (t.sync >= TENSOR_GPU_LAST_SYNC) {
		hipDeviceSynchronize();
		TENSOR_GPU_LAST_SYNC++;
	}
}

void tensor3_unsync(Tensor3 *t)
{
	t->sync = TENSOR_GPU_LAST_SYNC;
}

void tensor3_free(Tensor3 t) 
{
	tensor3_sync(t);
	free(t.cpu_data);
	hipFree(t.data);
}

void tensor3_copy_gpu_to_cpu (Tensor3 t) 
{
	tensor3_sync(t);
	size_t size = t.d1 * t.d2 * t.d3  * sizeof(float);
	hipMemcpy(t.cpu_data, t.data, size, hipMemcpyDeviceToHost);

}

void tensor3_copy_cpu_to_gpu (Tensor3 t) 
{
	tensor3_sync(t);
	size_t size = t.d1 * t.d2 * t.d3  * sizeof(float);
	hipMemcpy(t.data, t.cpu_data, size, hipMemcpyHostToDevice);
}

void tensor3_show(Tensor3 t) 
{
	tensor3_copy_gpu_to_cpu(t);
	printf("Tensor:\n");
	for (int k = 0; k < t.d3; k++) {
		printf("[ ");
		for (int j = 0; j < t.d2; j++) {
			printf("[ ");
			for (int i = 0; i < t.d1; i++) {
				printf("%.2f ", t.cpu_data[KEY(k, j, i, t.d3, t.d2, t.d1)]);
			}
			printf("] ");
		}
		printf("]\n");
	}
}

int int_ceil(int a, int b) 
{
	return (a + b-1) / b;
}

static const int BLOCK_SIZE = 32;

void vector_add(float *dest, float *a, float *b, int n)
{
	int n_block = int_ceil(n, BLOCK_SIZE);
	kernel_vector_add<<<n_block, BLOCK_SIZE>>>(dest, a, b, n);
}

void vector_mul(float *dest, float *a, float *b, int n)
{
	int n_block = int_ceil(n, BLOCK_SIZE);
	kernel_vector_mul<<<n_block, BLOCK_SIZE>>>(dest, a, b, n);
}

void vector_exp(float *dest, float *a, int n)
{
	int n_block = int_ceil(n, BLOCK_SIZE);
	kernel_vector_exp<<<n_block, BLOCK_SIZE>>>(dest, a, n);
}


Tensor3 tensor3_add(Tensor3 a, Tensor3 b) 
{
	tensor3_sync(a); tensor3_sync(b);
	assert(a.d1 == b.d1 && a.d2 == b.d2 && a.d3 == b.d3);
	Tensor3 c = tensor3_new(a.d1, a.d2, a.d3);
	vector_add(c.data, a.data, b.data, a.d1 * a.d2 * a.d3);
	tensor3_unsync(&c);
	return c;
}

Tensor3 tensor3_sum1(Tensor3 a) 
{
	tensor3_sync(a);
	Tensor3 c = tensor3_new(1, a.d2, a.d3);

	dim3 threads_per_block(1, BLOCK_SIZE/2, BLOCK_SIZE/2);
	dim3 num_blocks(1, int_ceil(c.d2, BLOCK_SIZE/2), int_ceil(c.d3, BLOCK_SIZE/2));

	kernel_sum<<<num_blocks, threads_per_block>>>(
		c.data, a.data, 
		c.d1, c.d2, c.d3, // c dimensions
		a.d1, a.d2, a.d3 // a dimensions
	);             

	tensor3_unsync(&c);
	return c;
}

bool tensor3_same_shape(Tensor3 a, Tensor3 b)
{
	return (a.d1 == b.d1 && a.d2 == b.d2 && a.d3 == b.d3);
}

bool tensor3_is_broadcastable(Tensor3 a, int d1, int d2, int d3)
{
	return ((a.d1 == d1 || a.d1 == 1) &&
			(a.d2 == d2 || a.d2 == 1) &&
			(a.d3 == d3 || a.d3 == 1));
}

bool tensor3_is_summable(Tensor3 a, int d1, int d2, int d3)
{
	return ((a.d1 == d1 || d1 == 1) &&
			(a.d2 == d2 || d2 == 1) &&
			(a.d3 == d3 || d3 == 1));
}

// Broadcast A tensor to dimensions (d1, d2, d3)
Tensor3 tensor3_broadcast(Tensor3 a, int d1, int d2, int d3) 
{
	assert(tensor3_is_broadcastable(a, d1, d2, d3));
	tensor3_sync(a);
	Tensor3 c = tensor3_new(d1, d2, d3);

	dim3 threads_per_block(THREAD_PER_BLOCK_X, THREAD_PER_BLOCK_Y, THREAD_PER_BLOCK_Z);
	dim3 num_blocks(
		int_ceil(c.d1, threads_per_block.x),
		int_ceil(c.d2, threads_per_block.y),
		int_ceil(c.d3, threads_per_block.z)
	);

	kernel_broadcast<<<num_blocks, threads_per_block>>>(
		c.data, a.data, 
		c.d1, c.d2, c.d3, // c dimensions
		a.d1, a.d2, a.d3  // a dimensions
	);             

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		ERROR("Kernel launch failed: %s", hipGetErrorString(err));
	}

	tensor3_unsync(&c);
	return c;
}

Tensor3 tensor3_sum(Tensor3 a, int d1, int d2, int d3) 
{
	assert(tensor3_is_summable(a, d1, d2, d3));
	tensor3_sync(a);
	Tensor3 c = tensor3_new(d1, d2, d3);

	dim3 threads_per_block(THREAD_PER_BLOCK_X, THREAD_PER_BLOCK_Y, THREAD_PER_BLOCK_Z);
	dim3 num_blocks(
		int_ceil(c.d1, threads_per_block.x),
		int_ceil(c.d2, threads_per_block.y),
		int_ceil(c.d3, threads_per_block.z)
	);

	kernel_sum<<<num_blocks, threads_per_block>>>(
		c.data, a.data, 
		c.d1, c.d2, c.d3, // c dimensions
		a.d1, a.d2, a.d3  // a dimensions
	);             

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		ERROR("Kernel launch failed: %s", hipGetErrorString(err));
	}

	tensor3_unsync(&c);
	return c;
}


Tensor3 tensor3_div(Tensor3 a, Tensor3 b) 
{
	tensor3_sync(a); tensor3_sync(b);
	Tensor3 c = tensor3_new(max(a.d1, b.d1), max(a.d2, b.d2), max(a.d3, b.d3));
	if (!tensor3_same_shape(a, b)) {
		if (tensor3_is_broadcastable(a, b.d1, b.d2, b.d3)) {
			a = tensor3_broadcast(a, b.d1, b.d2, b.d3);
		}
		if (tensor3_is_broadcastable(b, a.d1, a.d2, a.d3)) {
			b = tensor3_broadcast(b, a.d1, a.d2, a.d3);
		}
		else {
			ERROR("Shapes (%d, %d, %d) and (%d, %d, %d) are not broadcastable\n",
			      a.d1, a.d2, a.d3, b.d1, b.d2, b.d3);
		}
	}

	int n = c.d1 * c.d2 * c.d3;
	int n_block = int_ceil(n, BLOCK_SIZE);
	kernel_vector_div<<<n_block, BLOCK_SIZE>>>(c.data, a.data, b.data, n);
	tensor3_unsync(&c);
	return c;
}

Tensor3 tensor3_mul(Tensor3 a, Tensor3 b) 
{
	tensor3_sync(a); tensor3_sync(b);
	assert(a.d1 == b.d1 && a.d2 == b.d2 && a.d3 == b.d3);
	Tensor3 c = tensor3_new(a.d1, a.d2, a.d3);
	vector_mul(c.data, a.data, b.data, a.d1 * a.d2 * a.d3);
	tensor3_unsync(&c);
	return c;
}

Tensor3 tensor3_exp(Tensor3 a) 
{
	tensor3_sync(a);
	Tensor3 c = tensor3_new(a.d1, a.d2, a.d3);
	vector_exp(c.data, a.data, a.d1 * a.d2 * a.d3);
	tensor3_unsync(&c);
	return c;
}

Tensor3 tensor3_cpu_naive_matmul(Tensor3 a, Tensor3 b)
{
	tensor3_sync(a); tensor3_sync(b);
	assert(a.d1 == b.d2);
	int N = a.d2;
	int M = b.d1;
	int T = a.d1;
	Tensor3 c = tensor3_new(M, N, max(a.d3, b.d3));

	float *A = a.cpu_data;
	float *B = b.cpu_data;
	float *C = c.cpu_data;

	if (a.d3 == b.d3){
		if (a.d3 == 1) {
			for(size_t i = 0; i < N; i++) {
				for(size_t k = 0; k < T; k++) {
					for(size_t j = 0; j < M; j++) {
						C[i*M + j] += A[i*T + k] * B[k*M + j];
					}
				}
			}
		}
		else {
			TODO("Not implemented");
		}
	}
	else {
		TODO("Not implemented");
	}
	return c;
}

// TODO: implement more paralel version
Tensor3 tensor3_matmul(Tensor3 a, Tensor3 b) 
{
	tensor3_sync(a); tensor3_sync(b);
	assert(a.d1 == b.d2);
	assert(a.d3 == b.d3 || a.d3 == 1 || b.d3 == 1);
	int N = a.d2;
	int M = b.d1;
	int T = a.d1;
	Tensor3 c = tensor3_new(M, N, max(a.d3, b.d3));
	dim3 threads_per_block(BLOCK_SIZE/2, BLOCK_SIZE/2);
	dim3 num_blocks(int_ceil(M, BLOCK_SIZE/2), int_ceil(N, BLOCK_SIZE/2));
	for (int i = 0; i < c.d3; i++) {
		kernel_mat_mul<<<num_blocks, threads_per_block>>>(
			c.data + i*(c.d1 * c.d2), 
			a.data + i*(a.d1 * a.d2) * (a.d3 > 1), 
			b.data + i*(b.d1 * b.d2) * (b.d3 > 1), 
			N, M, T
		);
	}
	tensor3_unsync(&c);
	return c;
}
