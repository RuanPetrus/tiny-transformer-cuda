#include "hip/hip_runtime.h"
#include <assert.h>
#include <random>
#include "kernels.cu"

#define TODO(message) do { fprintf(stderr, "%s:%d: TODO: %s\n", __FILE__, __LINE__, message); abort(); } while(0)
#define ERROR(...) do { fprintf(stderr, __VA_ARGS__); abort(); } while(0)

#define THREAD_PER_BLOCK_X (1 << 4)
#define THREAD_PER_BLOCK_Y (1 << 3)
#define THREAD_PER_BLOCK_Z (1 << 3)

static int TENSOR_GPU_LAST_SYNC = 0;

float random_normal_distribution_float()
{
	static std::random_device random_device{};
	static std::mt19937 random_generator{random_device()};
	static std::normal_distribution random_normal_distribution{0.0f, 1.0f};
	return random_normal_distribution(random_generator);
}

int int_ceil(int a, int b) 
{
	return (a + b-1) / b;
}

struct Tensor3 
{
	int d1, d2, d3;
	float *data; // Gpu data
	float *cpu_data;
	int sync;
};

Tensor3 tensor3_new(int d1, int d2, int d3, float *v)  // (d3, d2, d1)
{
	size_t size = d1 * d2 * d3  * sizeof(float);
	float *cpu_data =(float *) malloc(size);
	float *data; hipMalloc(&data, size);
	if (v != NULL) {
		hipMemcpy(data, v, size, hipMemcpyHostToDevice);
	}
	return {
		d1, d2, d3,
		data,
		cpu_data,
		-1
	};
}

Tensor3 tensor3_new(int d1, int d2, int d3) 
{
	return tensor3_new(d1, d2, d3, NULL);
}

Tensor3 tensor3_randn(int d1, int d2, int d3) 
{

	size_t size = d1 * d2 * d3  * sizeof(float);
	float *data =(float *) malloc(size);
	for (int i = 0; i < d1*d2*d3; i++) {
		data[i] = random_normal_distribution_float();
	}
	return tensor3_new(d1, d2, d3, data);
}

void tensor3_sync(Tensor3 t) 
{
	if (t.sync >= TENSOR_GPU_LAST_SYNC) {
		hipDeviceSynchronize();
		TENSOR_GPU_LAST_SYNC++;
	}
}

void tensor3_unsync(Tensor3 *t)
{
	t->sync = TENSOR_GPU_LAST_SYNC;
}

void tensor3_free(Tensor3 t) 
{
	tensor3_sync(t);
	free(t.cpu_data);
	hipFree(t.data);
}

void tensor3_copy_gpu_to_cpu (Tensor3 t) 
{
	tensor3_sync(t);
	size_t size = t.d1 * t.d2 * t.d3  * sizeof(float);
	hipMemcpy(t.cpu_data, t.data, size, hipMemcpyDeviceToHost);

}

void tensor3_copy_cpu_to_gpu (Tensor3 t) 
{
	tensor3_sync(t);
	size_t size = t.d1 * t.d2 * t.d3  * sizeof(float);
	hipMemcpy(t.data, t.cpu_data, size, hipMemcpyHostToDevice);
}

void tensor3_show(Tensor3 t) 
{
	tensor3_copy_gpu_to_cpu(t);
	printf("Tensor (%d, %d, %d):\n", t.d1, t.d2, t.d3);
	for (int k = 0; k < t.d3; k++) {
		printf("[ ");
		for (int j = 0; j < t.d2; j++) {
			printf("[ ");
			for (int i = 0; i < t.d1; i++) {
				printf("%.4f ", t.cpu_data[KEY(k, j, i, t.d3, t.d2, t.d1)]);
			}
			printf("] ");
		}
		printf("]\n");
	}
}

bool tensor3_same_shape(Tensor3 a, Tensor3 b)
{
	return (a.d1 == b.d1 && a.d2 == b.d2 && a.d3 == b.d3);
}

bool tensor3_is_broadcastable(Tensor3 a, int d1, int d2, int d3)
{
	return ((a.d1 == d1 || a.d1 == 1) &&
			(a.d2 == d2 || a.d2 == 1) &&
			(a.d3 == d3 || a.d3 == 1));
}

bool tensor3_is_summable(Tensor3 a, int d1, int d2, int d3)
{
	return ((a.d1 == d1 || d1 == 1) &&
			(a.d2 == d2 || d2 == 1) &&
			(a.d3 == d3 || d3 == 1));
}

Tensor3 tensor3_broadcast(Tensor3 a, int d1, int d2, int d3) 
{
	assert(tensor3_is_broadcastable(a, d1, d2, d3));
	tensor3_sync(a);
	Tensor3 c = tensor3_new(d1, d2, d3);

	dim3 threads_per_block(THREAD_PER_BLOCK_X, THREAD_PER_BLOCK_Y, THREAD_PER_BLOCK_Z);
	dim3 num_blocks(
		int_ceil(c.d1, threads_per_block.x),
		int_ceil(c.d2, threads_per_block.y),
		int_ceil(c.d3, threads_per_block.z)
	);

	kernel_broadcast<<<num_blocks, threads_per_block>>>(
		c.data, a.data, 
		c.d1, c.d2, c.d3, // c dimensions
		a.d1, a.d2, a.d3  // a dimensions
	);             

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		ERROR("Kernel launch failed: %s", hipGetErrorString(err));
	}

	tensor3_unsync(&c);
	return c;
}

Tensor3 tensor3_sum(Tensor3 a, int d1, int d2, int d3) 
{
	assert(tensor3_is_summable(a, d1, d2, d3));
	tensor3_sync(a);
	Tensor3 c = tensor3_new(d1, d2, d3);

	dim3 threads_per_block(THREAD_PER_BLOCK_X, THREAD_PER_BLOCK_Y, THREAD_PER_BLOCK_Z);
	dim3 num_blocks(
		int_ceil(c.d1, threads_per_block.x),
		int_ceil(c.d2, threads_per_block.y),
		int_ceil(c.d3, threads_per_block.z)
	);

	kernel_sum<<<num_blocks, threads_per_block>>>(
		c.data, a.data, 
		c.d1, c.d2, c.d3, // c dimensions
		a.d1, a.d2, a.d3  // a dimensions
	);             

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		ERROR("Kernel launch failed: %s", hipGetErrorString(err));
	}

	tensor3_unsync(&c);
	return c;
}

// op(1) == (1, 2)
// op(2) == (1, 3)
// op(3) == (2, 3)
Tensor3 tensor3_transpose(Tensor3 a, int dx, int dy) 
{
	tensor3_sync(a);
	int op = dx ^ dy;
	int d1 = a.d1, d2 = a.d2, d3 = a.d3;
	if (op == 3) {
		SWAP(d1, d2, int);
	}
	else if (op == 2) {
		SWAP(d1, d3, int);
	}
	else if (op == 1) {
		SWAP(d2, d3, int);
	}
	Tensor3 c = tensor3_new(d1, d2, d3);

	dim3 threads_per_block(THREAD_PER_BLOCK_X, THREAD_PER_BLOCK_Y, THREAD_PER_BLOCK_Z);
	dim3 num_blocks(
		int_ceil(a.d1, threads_per_block.x),
		int_ceil(a.d2, threads_per_block.y),
		int_ceil(a.d3, threads_per_block.z)
	);

	kernel_transpose<<<num_blocks, threads_per_block>>>(
		c.data, a.data, 
		a.d1, a.d2, a.d3,
		op          
	);             

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		ERROR("Kernel launch failed: %s", hipGetErrorString(err));
	}

	tensor3_unsync(&c);
	return c;
}

Tensor3 tensor3_view(Tensor3 a, int d1, int d2, int d3) 
{
	assert(d1*d2*d3 == a.d1*a.d2*a.d3);
	tensor3_sync(a);
	Tensor3 c = tensor3_new(d1, d2, d3);

	hipMemcpy(c.data, a.data, d1*d2*d3*sizeof(float), hipMemcpyDeviceToDevice);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		ERROR("Kernel launch failed: %s", hipGetErrorString(err));
	}

	tensor3_unsync(&c);
	return c;
}

Tensor3 tensor3_add(Tensor3 a, Tensor3 b) 
{
	tensor3_sync(a); tensor3_sync(b);
	Tensor3 c = tensor3_new(max(a.d1, b.d1), max(a.d2, b.d2), max(a.d3, b.d3));
	if (!tensor3_same_shape(a, b)) {
		if (tensor3_is_broadcastable(a, b.d1, b.d2, b.d3)) {
			a = tensor3_broadcast(a, b.d1, b.d2, b.d3);
		}
		if (tensor3_is_broadcastable(b, a.d1, a.d2, a.d3)) {
			b = tensor3_broadcast(b, a.d1, a.d2, a.d3);
		}
		else {
			ERROR("Shapes (%d, %d, %d) and (%d, %d, %d) are not broadcastable\n",
			      a.d1, a.d2, a.d3, b.d1, b.d2, b.d3);
		}
	}

	dim3 threads_per_block(THREAD_PER_BLOCK_X, THREAD_PER_BLOCK_Y, THREAD_PER_BLOCK_Z);
	dim3 num_blocks(
		int_ceil(c.d1, threads_per_block.x),
		int_ceil(c.d2, threads_per_block.y),
		int_ceil(c.d3, threads_per_block.z)
	);

	kernel_add<<<num_blocks, threads_per_block>>>(
		c.data, a.data, b.data,
		c.d1, c.d2, c.d3
	);             

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		ERROR("Kernel launch failed: %s", hipGetErrorString(err));
	}

	tensor3_unsync(&c);
	return c;
}


Tensor3 tensor3_div(Tensor3 a, Tensor3 b) 
{
	tensor3_sync(a); tensor3_sync(b);
	Tensor3 c = tensor3_new(max(a.d1, b.d1), max(a.d2, b.d2), max(a.d3, b.d3));
	if (!tensor3_same_shape(a, b)) {
		if (tensor3_is_broadcastable(a, b.d1, b.d2, b.d3)) {
			a = tensor3_broadcast(a, b.d1, b.d2, b.d3);
		}
		if (tensor3_is_broadcastable(b, a.d1, a.d2, a.d3)) {
			b = tensor3_broadcast(b, a.d1, a.d2, a.d3);
		}
		else {
			ERROR("Shapes (%d, %d, %d) and (%d, %d, %d) are not broadcastable\n",
			      a.d1, a.d2, a.d3, b.d1, b.d2, b.d3);
		}
	}

	dim3 threads_per_block(THREAD_PER_BLOCK_X, THREAD_PER_BLOCK_Y, THREAD_PER_BLOCK_Z);
	dim3 num_blocks(
		int_ceil(c.d1, threads_per_block.x),
		int_ceil(c.d2, threads_per_block.y),
		int_ceil(c.d3, threads_per_block.z)
	);

	kernel_div<<<num_blocks, threads_per_block>>>(
		c.data, a.data, b.data,
		c.d1, c.d2, c.d3
	);             

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		ERROR("Kernel launch failed: %s", hipGetErrorString(err));
	}

	tensor3_unsync(&c);
	return c;
}

Tensor3 tensor3_mul(Tensor3 a, Tensor3 b) 
{
	tensor3_sync(a); tensor3_sync(b);
	Tensor3 c = tensor3_new(max(a.d1, b.d1), max(a.d2, b.d2), max(a.d3, b.d3));
	if (!tensor3_same_shape(a, b)) {
		if (tensor3_is_broadcastable(a, b.d1, b.d2, b.d3)) {
			a = tensor3_broadcast(a, b.d1, b.d2, b.d3);
		}
		if (tensor3_is_broadcastable(b, a.d1, a.d2, a.d3)) {
			b = tensor3_broadcast(b, a.d1, a.d2, a.d3);
		}
		else {
			ERROR("Shapes (%d, %d, %d) and (%d, %d, %d) are not broadcastable\n",
			      a.d1, a.d2, a.d3, b.d1, b.d2, b.d3);
		}
	}

	dim3 threads_per_block(THREAD_PER_BLOCK_X, THREAD_PER_BLOCK_Y, THREAD_PER_BLOCK_Z);
	dim3 num_blocks(
		int_ceil(c.d1, threads_per_block.x),
		int_ceil(c.d2, threads_per_block.y),
		int_ceil(c.d3, threads_per_block.z)
	);

	kernel_mul<<<num_blocks, threads_per_block>>>(
		c.data, a.data, b.data,
		c.d1, c.d2, c.d3
	);             

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		ERROR("Kernel launch failed: %s", hipGetErrorString(err));
	}

	tensor3_unsync(&c);
	return c;
}

Tensor3 tensor3_exp(Tensor3 a) 
{
	tensor3_sync(a);
	Tensor3 c = tensor3_new(a.d1, a.d2, a.d3);
	dim3 threads_per_block(THREAD_PER_BLOCK_X, THREAD_PER_BLOCK_Y, THREAD_PER_BLOCK_Z);
	dim3 num_blocks(
		int_ceil(c.d1, threads_per_block.x),
		int_ceil(c.d2, threads_per_block.y),
		int_ceil(c.d3, threads_per_block.z)
	);

	kernel_exp<<<num_blocks, threads_per_block>>>(
		c.data, a.data,
		c.d1, c.d2, c.d3
	);             

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		ERROR("Kernel launch failed: %s", hipGetErrorString(err));
	}

	tensor3_unsync(&c);
	return c;
}

Tensor3 tensor3_log(Tensor3 a) 
{
	tensor3_sync(a);
	Tensor3 c = tensor3_new(a.d1, a.d2, a.d3);
	dim3 threads_per_block(THREAD_PER_BLOCK_X, THREAD_PER_BLOCK_Y, THREAD_PER_BLOCK_Z);
	dim3 num_blocks(
		int_ceil(c.d1, threads_per_block.x),
		int_ceil(c.d2, threads_per_block.y),
		int_ceil(c.d3, threads_per_block.z)
	);

	kernel_log<<<num_blocks, threads_per_block>>>(
		c.data, a.data,
		c.d1, c.d2, c.d3
	);             

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		ERROR("Kernel launch failed: %s", hipGetErrorString(err));
	}

	tensor3_unsync(&c);
	return c;
}

Tensor3 tensor3_matmul(Tensor3 a, Tensor3 b) 
{
	tensor3_sync(a); tensor3_sync(b);
	assert(a.d1 == b.d2);
	Tensor3 c = tensor3_new(b.d1, a.d2, max(a.d3, b.d3));
	if (b.d3 != a.d3) {
		if (a.d3 == 1) {
			a = tensor3_broadcast(a, a.d1, a.d2, b.d3);
		}
		if (b.d3 == 1) {
			b = tensor3_broadcast(b, b.d1, b.d2, a.d3);
		}
		else {
			ERROR("Shapes (%d, %d, %d) and (%d, %d, %d) are not broadcastable\n",
			      a.d1, a.d2, a.d3, b.d1, b.d2, b.d3);
		}
	}

	dim3 threads_per_block(THREAD_PER_BLOCK_X, THREAD_PER_BLOCK_Y, THREAD_PER_BLOCK_Z);
	dim3 num_blocks(
		int_ceil(c.d1, threads_per_block.x),
		int_ceil(c.d2, threads_per_block.y),
		int_ceil(c.d3, threads_per_block.z)
	);

	kernel_matmul<<<num_blocks, threads_per_block>>>(
		c.data, a.data, b.data,
		c.d1, c.d2, c.d3,
		a.d1
	);             

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		ERROR("Kernel launch failed: %s", hipGetErrorString(err));
	}

	tensor3_unsync(&c);
	return c;
}

Tensor3 tensor3_mul_scalar(Tensor3 a, float b) 
{
	tensor3_sync(a);
	Tensor3 c = tensor3_new(a.d1, a.d2, a.d3);

	dim3 threads_per_block(THREAD_PER_BLOCK_X, THREAD_PER_BLOCK_Y, THREAD_PER_BLOCK_Z);
	dim3 num_blocks(
		int_ceil(c.d1, threads_per_block.x),
		int_ceil(c.d2, threads_per_block.y),
		int_ceil(c.d3, threads_per_block.z)
	);

	kernel_mul_scalar<<<num_blocks, threads_per_block>>>(
		c.data, a.data, b,
		c.d1, c.d2, c.d3
	);             

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		ERROR("Kernel launch failed: %s", hipGetErrorString(err));
	}

	tensor3_unsync(&c);
	return c;
}

Tensor3 tensor3_one_hot_cpu(int *arr, int B, int T, int A)
{
	float *res = (float *) malloc(B * T * A * sizeof(float));
	for (int k = 0; k < B; k++)
	for (int j = 0; j < T; j++)
	for (int i = 0; i < A; i++) {
		res[KEY(k, j, i, B, T, A)] = (arr[KEY(0, k, j, 0, B, T)] == i ? 1.0 : 0.0);
	}
	return tensor3_new(A, T, B, res);
}
